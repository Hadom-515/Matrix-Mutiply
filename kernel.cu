
#include <hip/hip_runtime.h>
#include <stdio.h>

#define TILE_SIZE 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     * Use shared memory for tiling
     *
     ********************************************************************/

    /*************************************************************************/
    // INSERT KERNEL CODE HERE
  float CValue = 0;

    int Row = blockIdx.y*TILE_SIZE + threadIdx.y;
    int Col = blockIdx.x*TILE_SIZE + threadIdx.x;

    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    for (int i = 0; i <  (k-1)/(float)TILE_SIZE+1; i++) {

         if (i*TILE_SIZE + threadIdx.x < k && Row < m)
             sA[threadIdx.y][threadIdx.x] = A[Row*k + i*TILE_SIZE + threadIdx.x];
         else
             sA[threadIdx.y][threadIdx.x] = 0;

         if (i*TILE_SIZE + threadIdx.y < k && Col < n)
             sB[threadIdx.y][threadIdx.x] = B[(i*TILE_SIZE + threadIdx.y)*n + Col];
         else
             sB[threadIdx.y][threadIdx.x] = 0;

         __syncthreads();

         for (int n = 0; n < TILE_SIZE; ++n)
             CValue += sA[threadIdx.y][n] * sB[n][threadIdx.x];

         __syncthreads();
    }

    if (Row < m && Col < n)
        C[Row*n+Col] = CValue;
    /*************************************************************************/
}

void basicSgemm(int m, int n, int k, const float *A, const float *B, float *C)
{
    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = TILE_SIZE;
	
    /*************************************************************************/
    //INSERT CODE HERE
    dim3 DimGrid((n-1)/BLOCK_SIZE+1,(m-1)/BLOCK_SIZE+1,1);
    dim3 DimBlock(BLOCK_SIZE,BLOCK_SIZE,1);

    /*************************************************************************/

    // Invoke CUDA kernel -----------------------------------------------------

    /*************************************************************************/
    //INSERT CODE HERE
	mysgemm<<<DimGrid,DimBlock>>>(m,n,k,A,B,C);
    /*************************************************************************/
}


